#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "../include/ParSecureML.h"
void MallocD(float *&gpu_a, int size){
    hipError_t cudaStat;
    cudaStat = hipMalloc((void**)&gpu_a, sizeof(*gpu_a)*size);
    if(cudaStat != hipSuccess){
        cout << "Malloc failed:" << hipGetErrorString(cudaStat)<< endl;
        exit(0);
    }
}
void CopyHtoD(float *gpu_a, float *a, int size){
    hipError_t cudaStat;
    cudaStat = hipMemcpy(gpu_a, a, sizeof(*a)*size, hipMemcpyHostToDevice);
    if(cudaStat != hipSuccess){
        cout << "Error code:" << cudaStat << endl;
        cout << "CopyHtoD failed." << endl;
        exit(0);
    }
}
void CopyDtoH(float *&a, float *&gpu_a, int size){
    hipError_t cudaStat;
    cudaStat = hipMemcpy(a, gpu_a, sizeof(*a)*size, hipMemcpyDeviceToHost);
    if(cudaStat != hipSuccess){
        cout << "Error code:" << cudaStat << endl;
        cout << "CopyDtoH failed." << endl;
        exit(0);
    }
}
void Support::GPU_Mul(){
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        cout << "CUBLAS create failed." << endl;
        exit(0);
    }
    float alpha = 1;
    float b = 0;
    
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row1, col2, col1, &alpha, GPU_U, row1, GPU_V, row2, &b, GPU_Z, row1);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        cout << "Cublas sgemm failed." << endl;
        exit(0);
    }
}
void ReleaseGPU(float *A){
    hipFree(A);
}
__global__ void cudaTripletSum(float *sum, float *fac1, float *fac2, float *fac3,  int size){
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int cur = bid*blockDim.x+tid;
    if(cur >= size) return;
    float tmp = fac1[cur] + fac2[cur] + fac3[cur];
    sum[cur] = tmp;
}
__global__ void cudaSum(float *A, float *B, float *sum, int size){
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int cur = bid*blockDim.x+tid;
    if(cur >= size) return;
    float tmp = A[cur]+B[cur];
    sum[cur] = tmp;
}
__global__ void cudaMinus(float *A, float *B, float *min, int size){
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int cur = bid*blockDim.x+tid;
    if(cur >= size) return;
    float tmp = A[cur]-B[cur];
    min[cur] = tmp;
}
void Triplet::cudaTripletMul(int flag){
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipError_t cudaStat;
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        cout << "CUBLAS create failed." << endl;
        exit(0);
    }
    float alpha1 = 1;
    float alpha2 = 1;
    float b = 0;
    while(flag1 == 0){
        continue;
    }
    if(flag == 0){
        while(flag2 == 0){
            continue;
        }
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row1, col2, col1, &alpha1, GPU_A, row1, GPU_F, row2, &b, fac1, row1);
        if(stat != HIPBLAS_STATUS_SUCCESS){
            cout << "Cublas sgemm failed." << endl;
            exit(0);
        }
    }
    else if(flag == 1){
        cudaMinus<<<row1*col1/1024+1, 1024>>>(GPU_A, GPU_E, GPU_D, row1*col1);
        while(flag2 == 0){
            continue;
        }
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row1, col2, col1, &alpha1, GPU_D, row1, GPU_F, row2, &b, fac1, row1);
        if(stat != HIPBLAS_STATUS_SUCCESS){
            cout << "Cublas sgemm failed." << endl;
            exit(0);
        }
    }
    while(flag3 == 0){
        continue;
    }
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row1, col2, col1, &alpha2, GPU_E, row1, GPU_B, row2, &b, fac2, row1);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        cout << "Cublas sgemm failed." << endl;
        exit(0);
    }
    hipblasDestroy(handle);
    cudaTripletSum<<<row1*col1/1024+1024, 1024>>>(GPU_C, fac1, fac2, GPU_Z, row1*col2);
    cudaStat = hipGetLastError();
    if(cudaStat != hipSuccess){
        cout << "Kernel launch failed." << endl;
        exit(0);
    }
}

__global__ void cudaConv(int flag, float *GPU_A, float *GPU_B, float *GPU_C, float *GPU_E, float *GPU_F, float *GPU_Z, int row1, int col1, int row2, int col2, int o_row, int o_col, int num){
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int cur = bid*blockDim.x+tid;
    if(cur >= num*o_row*o_col) return;
    int num_cur = cur/(o_row*o_col);
    int row_cur = cur%(o_row*o_col)/o_col;
    int col_cur = cur%(o_row*o_col)%o_col;
    float tem = 0;
    for(int i = 0; i < row2; i++){
        for(int j = 0; j < col2; j++){
            tem += flag*GPU_E[num_cur*o_row*o_col*row2*col2+row_cur*o_col*row2*col2+col_cur*row2*col2+i*col2*j]*GPU_F[i*col2+j] + GPU_A[num_cur*row1*col1+(row_cur+i)*row1+col_cur+j] * GPU_F[i*col2+j] + GPU_E[num_cur*o_row*o_col*row2*col2+row_cur*o_col*row2*col2+col_cur*row2*col2+i*col2*j] * GPU_B[i*col2+j] + GPU_Z[i*col2+j];
        }
    }
    GPU_C[num_cur*o_row*o_col+row_cur*o_col+col_cur] = tem;
    
}
void ConvTriplet::GPU_OP(int flag){
    cudaConv<<<256, 256>>>(flag, GPU_A, GPU_B, GPU_C, GPU_E, GPU_F, GPU_Z, row1, col1, row2, col2, o_row, o_row, num);
}